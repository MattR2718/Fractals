#include "hip/hip_runtime.h"
#include "mandelbrot.cuh"

std::vector<double> linspace(double start, double end, int num)
{
  std::vector<double> linspaced;
  if (num == 0) { return linspaced; }
  if (num == 1) 
    {
      linspaced.push_back(start);
      return linspaced;
    }
  double delta = (end - start) / (num - 1);
  for(int i=0; i < num-1; ++i)
    {
      linspaced.push_back(start + delta * i);
    }
  linspaced.push_back(end);
  return linspaced;
}

__global__ void mandel(int n, const int maxIter, thrust::complex<double> *v, int *numIterations){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < n; i+=stride){
        thrust::complex<double> z = thrust::complex<double>(0.0, 0.0);
        int iters = 0;
        for (iters = 0; iters < maxIter; iters++){
            z = z * z + v[i];
            if (thrust::abs(z) > 4){
                numIterations[i] = iters;
                break;
            }
            numIterations[i] = maxIter;
        }
        
    }
}

void mandelbrot(const int& width, const int& height, std::pair<double, double> tl, std::pair<double, double> br, const int& maxIter, int *numIterations){
    int N = width * height;
    thrust::complex<double> *v;
    int *numIt;
    hipMallocManaged(&numIt, N*sizeof(int));
    hipMallocManaged(&v, N*sizeof(thrust::complex<double>));

    std::vector<double> xvalues = linspace(tl.first, br.first, width);
    std::vector<double> yvalues = linspace(tl.second, br.second, height);
    for (int i = 0; i < xvalues.size(); i++){
        for (int j = 0; j < yvalues.size(); j++){
            v[(j * width) + i] = thrust::complex<double>(xvalues[i], yvalues[j]);
        }
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    mandel<<<numBlocks, blockSize>>>(N, maxIter, v, numIt);
    hipDeviceSynchronize();

    for (int i = 0; i < N; i++){
        numIterations[i] = numIt[i];
    }

    hipFree(numIt);
}
